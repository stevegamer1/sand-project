#include "hip/hip_runtime.h"
#include <iostream>
#include "sand_calculator.hpp"


void checkErr() {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cout << "CUDA ERROR: " << hipGetErrorString(err) << "\n";
    }
}

__global__ void copy_low_row_CUDA(float* input, float* result) {
    int x = 1 + blockDim.x * blockIdx.x + threadIdx.x;
    result[x] = input[x];
}

__global__ void sand_kernel_CUDA(float* input, float* result) {
    int x = 1 + blockDim.x * blockIdx.x + threadIdx.x;
    int y = 1 + blockDim.y * blockIdx.y + threadIdx.y;
    if (input[y * 160 + x] != 0.0) {
        if (input[(y - 1) * 160 + x] == 0.0) {
            result[(y - 1) * 160 + x] = 1.0;
        } else if (input[(y - 1) * 160 + x - 1] == 0.0) {
            result[(y - 1) * 160 + x - 1] = 1.0;
        } else if (input[(y - 1) * 160 + x + 1] == 0.0) {
            result[(y - 1) * 160 + x + 1] = 1.0;
        } else {
            result[y * 160 + x] = 1.0;
        }
    }
}


void calculate_sand_CUDA(float* host_input) {
    float* input;
    float* result;
    hipMalloc(&input, 4 * 160 * 120);
    checkErr();
    hipMalloc(&result, 4 * 160 * 120);
    checkErr();
    hipMemcpy(input, host_input, 4 * 160 * 120, hipMemcpyHostToDevice);
    checkErr();

    copy_low_row_CUDA<<<158, 1>>>(input, result);
    checkErr();
    sand_kernel_CUDA<<<158, 119>>>(input, result);
    checkErr();
    hipDeviceSynchronize();
    checkErr();

    hipMemcpy(host_input, result, 4 * 160 * 120, hipMemcpyDeviceToHost);
    checkErr();
    hipFree(input);
    checkErr();
    hipFree(result);
    checkErr();
}
